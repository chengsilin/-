#include "hip/hip_runtime.h"
//nvcc -arch=sm_10 -o parallel_cuda parallel_cuda.cu `pkg-config --libs --cflags cudaopencv`
#include <iostream>
//#include <opencv2/opencv.hpp>
#include <opencv/highgui.h>
#include <opencv/cv.h>
//#include <opencv2/imgproc/imgproc.hpp>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"  
#include "" 

using namespace cv;
using namespace std;

double get_time()//获取当前时间
{
	double time[2];	
	struct timeval time_tmp;
	gettimeofday(&time_tmp, NULL);
	time[0]=time_tmp.tv_sec;//秒数
	time[1]=time_tmp.tv_usec;//微秒数
	return time[0]+time[1]*1.0e-6;
}

__global__ void parallel_cuda(uchar *dev_src,uchar *dev_dst,int row,int col,int NUM,int channel)
{
	int thread_id1 = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
	int block_row = row / NUM + 1;
	int i, j, row_start, row_end;
	if(thread_id1 < NUM){
		row_start = thread_id1 * block_row + 1;
		row_end = row_start + block_row;
		
		for(i = row_start; (i < row_end)&&(i<row-1) ;i++){
			const uchar *pre = dev_src+(i-1)*channel*col;
			const uchar *cur = dev_src+(i)*channel*col;
			const uchar *next = dev_src+(i+1)*channel*col;
			uchar *outData = dev_dst+(i)*channel*col;
			for (j = channel; j < (col-1)*channel; j++) {
				int tmp = ( cur[j]*9 - pre[j- channel]- pre[j]- pre[j + channel]-cur[j- channel]-cur[j + channel] -next[j- channel]-next[j]-next[j + channel]);
				outData[j] = tmp;
			}
		}
	}
}

int main(void)
{
	Mat imagein = imread("lena.jpg");
	Mat imageout;
	imageout.create(imagein.size(),imagein.type());
	if( imagein.empty()==true)
	{
		printf( "Can't open file!\n " );
		return -1;
	}
	const int channel = imagein.channels();

	double time1, time2;
	time1 = get_time();
	dim3 grid(1, 1, 1);
	dim3 block(8, 8, 1);
	int thread_num = 8*8;
	int IMG_SIZE = sizeof(uchar)*imagein.rows * channel * imagein.cols;
	uchar *dev_dst;
	uchar *dev_src;
	hipError_t error = hipSuccess;
    error = hipMalloc((void **)&dev_dst, IMG_SIZE);
    error = hipMalloc((void **)&dev_src, IMG_SIZE);//GPU内空间分配
	if (error != hipSuccess) {
        printf("Fail to hipMalloc on GPU");
        return 1;
    }
	hipMemcpy(dev_src, imagein.ptr<uchar>(0), IMG_SIZE, hipMemcpyHostToDevice);//简单的数据转移
	parallel_cuda<<<grid, block>>>(dev_src,dev_dst,imagein.rows,imagein.cols,thread_num,channel);//对每一项计算
	hipDeviceSynchronize();
	hipMemcpy(imageout.ptr<uchar>(0), dev_dst, IMG_SIZE, hipMemcpyDeviceToHost);
	time2 =get_time();
	double runTime = (double)(time2 - time1);
	printf("Image convolution time in cuda: %lf \n",runTime/300);
    IplImage tmp = IplImage(imageout);
    CvArr* arr = (CvArr*)&tmp;	
	cvSaveImage("output.jpg" , arr);
	hipFree(dev_src);
	hipFree(dev_dst);
	return 0;
}
