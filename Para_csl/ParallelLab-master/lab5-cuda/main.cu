#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime_api.h>

#include "../morph-linear/morphlib.h"

__global__ void dilation(char *src, char*dst, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height)
		return;

	int set = 0;
	for (int i = 0; i < 9; ++i) {
		int x2 = x + (i / 3) - 1;
		int y2 = y + (i % 3) - 1;
		if (x2 >= 0 && y2 >= 0 && x2 < width && y2 < height) {
			int idx2 = y2 * width + x2;
			if (src[idx2]) {
				set = 1;
			}
		}
	}
	
	int idx = y * width + x;
	dst[idx] = set;//set;
}

int ceil_div(int a, int b) {
	return a % b == 0 ? a / b : a / b + 1;
}

void apply_morph(morph::BinaryImage& src, morph::BinaryImage& dst) {
	char *devSrc, *devDst;
	size_t size = src.width * src.height;
	hipMalloc((void**) &devSrc, size);
	hipMalloc((void**) &devDst, size);

	dim3 threadsPerBlock = dim3(32, 32);
	//dim3 blocksPerGrid = dim3(1);
	dim3 blocksPerGrid = dim3(ceil_div(src.width, 32), ceil_div(src.height, 32));

	hipMemcpy(devSrc, src.bytes, size, hipMemcpyHostToDevice);

	dilation<<<blocksPerGrid, threadsPerBlock>>>(devSrc, devDst, src.width, src.height);

	hipMemcpy(dst.bytes, devDst, size, hipMemcpyDeviceToHost);

	hipFree(devSrc);
	hipFree(devDst);
}

int main(int argc, char** argv) {
	char* temp;
	hipMalloc((void**) &temp, 16);
	hipFree(temp);

	morph::exec_test(argc, argv, apply_morph);
}