#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ void VecAdd(int* A, int* B, int* C, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {
		A[i] = B[i] + C[i];
	}
}

constexpr int N = 1024;

int A[N], B[N], C[N];

int main() {
	for (int i = 0; i < N; ++i) {
		B[i] = i + 1;
		C[i] = 2 * i - 3;
	}

	int *dstA, *dstB, *dstC;
	hipMalloc((void**) &dstA, N * sizeof(int));
	hipMalloc((void**) &dstB, N * sizeof(int));
	hipMalloc((void**) &dstC, N * sizeof(int));
	
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	hipMemcpy(dstB, B, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dstC, C, sizeof(int) * N, hipMemcpyHostToDevice);

	// Invoke kernel
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(dstA, dstB, dstC, N);
	
	// Copy mem from device to host
	hipMemcpy(A, dstA, N * sizeof(int), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(dstA);
	hipFree(dstB);
	hipFree(dstC);

	// Print result
	printf("Results: ");
	for (int i = 0; i < N; ++i) {
		printf("%d ", A[i]);
	}
	printf("\n");
}